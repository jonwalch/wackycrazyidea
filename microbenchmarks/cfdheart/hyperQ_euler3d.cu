#include "hip/hip_runtime.h"
// Copyright 2009, Andrew Corrigan, acorriga@gmu.edu
// This code is from the AIAA-2009-4001 paper

//#include <cutil.h>
#include <hip/hip_runtime_api.h>
#include <helper_timer.h>
#include <iostream>
#include <fstream>


//heartwall
#include <avilib.h>
#include <avimod.h>
#include "define.c"
params_common_change common_change;
__constant__ params_common_change d_common_change;
params_common common;
__constant__ params_common d_common;
params_unique unique[ALL_POINTS];								// cannot determine size dynamically so choose more than usually needed
__constant__ params_unique d_unique[ALL_POINTS];

 
/*
 * Options 
 * 
 */ 
#define GAMMA 1.4f
#define iterations 2000
#ifndef block_length
	#define block_length 256
#endif

#define NDIM 3
#define NNB 4

#define RK 3	// 3rd order RK
#define ff_mach 1.2f
#define deg_angle_of_attack 0.0f



#define VAR_DENSITY 0
#define VAR_MOMENTUM  1
#define VAR_DENSITY_ENERGY (VAR_MOMENTUM+NDIM)
#define NVAR (VAR_DENSITY_ENERGY+1)


/*
 * Generic functions
 */
template <typename T>
T* alloc(int N)
{
	T* t;
	checkCudaErrors(hipMalloc((void**)&t, sizeof(T)*N));
	return t;
}

template <typename T>
void dealloc(T* array)
{
	checkCudaErrors(hipFree((void*)array));
}

template <typename T>
void copy(T* dst, T* src, int N)
{
	checkCudaErrors(hipMemcpy((void*)dst, (void*)src, N*sizeof(T), hipMemcpyDeviceToDevice));
}

template <typename T>
void upload(T* dst, T* src, int N)
{
	checkCudaErrors(hipMemcpy((void*)dst, (void*)src, N*sizeof(T), hipMemcpyHostToDevice));
}

/*
 * Element-based Cell-centered FVM solver functions
 */
__constant__ float ff_variable[NVAR];
__constant__ float3 ff_flux_contribution_momentum_x[1];
__constant__ float3 ff_flux_contribution_momentum_y[1];
__constant__ float3 ff_flux_contribution_momentum_z[1];
__constant__ float3 ff_flux_contribution_density_energy[1];

__global__ void cuda_initialize_variables(int nelr, float* variables)
{
	const int i = (blockDim.x*blockIdx.x + threadIdx.x);
	for(int j = 0; j < NVAR; j++)
		variables[i + j*nelr] = ff_variable[j];
}
void initialize_variables(int nelr, float* variables)
{
	dim3 Dg(nelr / block_length), Db(block_length);
	cuda_initialize_variables<<<Dg, Db>>>(nelr, variables);
	getLastCudaError("initialize_variables failed");
}

__device__ __host__ inline void compute_flux_contribution(float& density, float3& momentum, float& density_energy, float& pressure, float3& velocity, float3& fc_momentum_x, float3& fc_momentum_y, float3& fc_momentum_z, float3& fc_density_energy)
{
	fc_momentum_x.x = velocity.x*momentum.x + pressure;
	fc_momentum_x.y = velocity.x*momentum.y;
	fc_momentum_x.z = velocity.x*momentum.z;
	
	
	fc_momentum_y.x = fc_momentum_x.y;
	fc_momentum_y.y = velocity.y*momentum.y + pressure;
	fc_momentum_y.z = velocity.y*momentum.z;

	fc_momentum_z.x = fc_momentum_x.z;
	fc_momentum_z.y = fc_momentum_y.z;
	fc_momentum_z.z = velocity.z*momentum.z + pressure;

	float de_p = density_energy+pressure;
	fc_density_energy.x = velocity.x*de_p;
	fc_density_energy.y = velocity.y*de_p;
	fc_density_energy.z = velocity.z*de_p;
}

__device__ inline void compute_velocity(float& density, float3& momentum, float3& velocity)
{
	velocity.x = momentum.x / density;
	velocity.y = momentum.y / density;
	velocity.z = momentum.z / density;
}
	
__device__ inline float compute_speed_sqd(float3& velocity)
{
	return velocity.x*velocity.x + velocity.y*velocity.y + velocity.z*velocity.z;
}

__device__ inline float compute_pressure(float& density, float& density_energy, float& speed_sqd)
{
	return (float(GAMMA)-float(1.0f))*(density_energy - float(0.5f)*density*speed_sqd);
}

__device__ inline float compute_speed_of_sound(float& density, float& pressure)
{
	return sqrtf(float(GAMMA)*pressure/density);
}


//===============================================================================================================================================================================================================
//===============================================================================================================================================================================================================
//	KERNEL FUNCTION
//===============================================================================================================================================================================================================
//===============================================================================================================================================================================================================

__device__ void heartwall_kernel() {

	//======================================================================================================================================================
	//	COMMON VARIABLES
	//======================================================================================================================================================

	/*__shared__ volatile int smem[1024];*/

	fp* d_in;
	int rot_row;
	int rot_col;
	int in2_rowlow;
	int in2_collow;
	int ic;
	int jc;
	int jp1;
	int ja1, ja2;
	int ip1;
	int ia1, ia2;
	int ja, jb;
	int ia, ib;
	float s;
	int i;
	int j;
	int row;
	int col;
	int ori_row;
	int ori_col;
	int position;
	float sum;
	int pos_ori;
	float temp;
	float temp2;
	int location;
	int cent;
	int tMask_row; 
	int tMask_col;
	float largest_value_current = 0;
	float largest_value = 0;
	int largest_coordinate_current = 0;
	int largest_coordinate = 0;
	float fin_max_val = 0;
	int fin_max_coo = 0;
	int largest_row;
	int largest_col;
	int offset_row;
	int offset_col;
	__shared__ float in_partial_sum[51];															// WATCH THIS !!! HARDCODED VALUE
	__shared__ float in_sqr_partial_sum[51];															// WATCH THIS !!! HARDCODED VALUE
	__shared__ float in_final_sum;
	__shared__ float in_sqr_final_sum;
	float mean;
	float mean_sqr;
	float variance;
	float deviation;
	__shared__ float denomT;
	__shared__ float par_max_val[131];															// WATCH THIS !!! HARDCODED VALUE
	__shared__ int par_max_coo[131];															// WATCH THIS !!! HARDCODED VALUE
	int pointer;
	__shared__ float d_in_mod_temp[2601];
	int ori_pointer;
	int loc_pointer;

	//======================================================================================================================================================
	//	THREAD PARAMETERS
	//======================================================================================================================================================

	int bx = blockIdx.x;																// get current horizontal block index (0-n)
	int tx = threadIdx.x;																// get current horizontal thread index (0-n)
	int ei_new;

	/*smem[threadIdx.x] = 0;*/

	//===============================================================================================================================================================================================================
	//===============================================================================================================================================================================================================
	//	GENERATE TEMPLATE
	//===============================================================================================================================================================================================================
	//===============================================================================================================================================================================================================

	// generate templates based on the first frame only
	if(d_common_change.frame_no == 0){

		//======================================================================================================================================================
		// GET POINTER TO TEMPLATE FOR THE POINT
		//======================================================================================================================================================

		// pointers to: current template for current point
		d_in = &d_unique[bx].d_T[d_unique[bx].in_pointer];

		//======================================================================================================================================================
		//	UPDATE ROW LOC AND COL LOC
		//======================================================================================================================================================

		// uptade temporary endo/epi row/col coordinates (in each block corresponding to point, narrow work to one thread)
		ei_new = tx;
		if(ei_new == 0){

			// update temporary row/col coordinates
			pointer = d_unique[bx].point_no*d_common.no_frames+d_common_change.frame_no;
			d_unique[bx].d_tRowLoc[pointer] = d_unique[bx].d_Row[d_unique[bx].point_no];
			d_unique[bx].d_tColLoc[pointer] = d_unique[bx].d_Col[d_unique[bx].point_no];

		}

		//======================================================================================================================================================
		//	CREATE TEMPLATES
		//======================================================================================================================================================

		// work
		ei_new = tx;
		while(ei_new < d_common.in_elem){

			// figure out row/col location in new matrix
			row = (ei_new+1) % d_common.in_rows - 1;												// (0-n) row
			col = (ei_new+1) / d_common.in_rows + 1 - 1;											// (0-n) column
			if((ei_new+1) % d_common.in_rows == 0){
				row = d_common.in_rows - 1;
				col = col-1;
			}

			// figure out row/col location in corresponding new template area in image and give to every thread (get top left corner and progress down and right)
			ori_row = d_unique[bx].d_Row[d_unique[bx].point_no] - 25 + row - 1;
			ori_col = d_unique[bx].d_Col[d_unique[bx].point_no] - 25 + col - 1;
			ori_pointer = ori_col*d_common.frame_rows+ori_row;

			// update template
			d_in[col*d_common.in_rows+row] = d_common_change.d_frame[ori_pointer];

			// go for second round
			ei_new = ei_new + NUMBER_THREADS;

		}

	}

	//===============================================================================================================================================================================================================
	//===============================================================================================================================================================================================================
	//	PROCESS POINTS
	//===============================================================================================================================================================================================================
	//===============================================================================================================================================================================================================

	// process points in all frames except for the first one
	if(d_common_change.frame_no != 0){

		//======================================================================================================================================================
		//	SELECTION
		//======================================================================================================================================================

		in2_rowlow = d_unique[bx].d_Row[d_unique[bx].point_no] - d_common.sSize;													// (1 to n+1)
		in2_collow = d_unique[bx].d_Col[d_unique[bx].point_no] - d_common.sSize;

		// work
		ei_new = tx;
		while(ei_new < d_common.in2_elem){

			// figure out row/col location in new matrix
			row = (ei_new+1) % d_common.in2_rows - 1;												// (0-n) row
			col = (ei_new+1) / d_common.in2_rows + 1 - 1;											// (0-n) column
			if((ei_new+1) % d_common.in2_rows == 0){
				row = d_common.in2_rows - 1;
				col = col-1;
			}

			// figure out corresponding location in old matrix and copy values to new matrix
			ori_row = row + in2_rowlow - 1;
			ori_col = col + in2_collow - 1;
			d_unique[bx].d_in2[ei_new] = d_common_change.d_frame[ori_col*d_common.frame_rows+ori_row];

			// go for second round
			ei_new = ei_new + NUMBER_THREADS;

		}

		//======================================================================================================================================================
		//	SYNCHRONIZE THREADS
		//======================================================================================================================================================

		__syncthreads();

		//======================================================================================================================================================
		//	CONVOLUTION
		//======================================================================================================================================================

		//====================================================================================================
		//	ROTATION
		//====================================================================================================

		// variables
		d_in = &d_unique[bx].d_T[d_unique[bx].in_pointer];

		// work
		ei_new = tx;
		while(ei_new < d_common.in_elem){

			// figure out row/col location in padded array
			row = (ei_new+1) % d_common.in_rows - 1;												// (0-n) row
			col = (ei_new+1) / d_common.in_rows + 1 - 1;											// (0-n) column
			if((ei_new+1) % d_common.in_rows == 0){
				row = d_common.in_rows - 1;
				col = col-1;
			}
		
			// execution
			rot_row = (d_common.in_rows-1) - row;
			rot_col = (d_common.in_rows-1) - col;
			d_in_mod_temp[ei_new] = d_in[rot_col*d_common.in_rows+rot_row];

			// go for second round
			ei_new = ei_new + NUMBER_THREADS;

		}

		//====================================================================================================
		//	SYNCHRONIZE THREADS
		//====================================================================================================

		__syncthreads();

		//====================================================================================================
		//	ACTUAL CONVOLUTION
		//====================================================================================================

		// work
		ei_new = tx;
		while(ei_new < d_common.conv_elem){

			// figure out row/col location in array
			ic = (ei_new+1) % d_common.conv_rows;												// (1-n)
			jc = (ei_new+1) / d_common.conv_rows + 1;											// (1-n)
			if((ei_new+1) % d_common.conv_rows == 0){
				ic = d_common.conv_rows;
				jc = jc-1;
			}

			//
			j = jc + d_common.joffset;
			jp1 = j + 1;
			if(d_common.in2_cols < jp1){
				ja1 = jp1 - d_common.in2_cols;
			}
			else{
				ja1 = 1;
			}
			if(d_common.in_cols < j){
				ja2 = d_common.in_cols;
			}
			else{
				ja2 = j;
			}

			i = ic + d_common.ioffset;
			ip1 = i + 1;
			
			if(d_common.in2_rows < ip1){
				ia1 = ip1 - d_common.in2_rows;
			}
			else{
				ia1 = 1;
			}
			if(d_common.in_rows < i){
				ia2 = d_common.in_rows;
			}
			else{
				ia2 = i;
			}

			s = 0;

			for(ja=ja1; ja<=ja2; ja++){
				jb = jp1 - ja;
				for(ia=ia1; ia<=ia2; ia++){
					ib = ip1 - ia;
					s = s + d_in_mod_temp[d_common.in_rows*(ja-1)+ia-1] * d_unique[bx].d_in2[d_common.in2_rows*(jb-1)+ib-1];
				}
			}

			//d_unique[bx].d_conv[d_common.conv_rows*(jc-1)+ic-1] = s;
			d_unique[bx].d_conv[ei_new] = s;

			// go for second round
			ei_new = ei_new + NUMBER_THREADS;

		}

		//======================================================================================================================================================
		//	SYNCHRONIZE THREADS
		//======================================================================================================================================================

		__syncthreads();

		//======================================================================================================================================================
		//	CUMULATIVE SUM
		//======================================================================================================================================================

		//====================================================================================================
		//	PAD ARRAY, VERTICAL CUMULATIVE SUM
		//====================================================================================================

		//==================================================
		//	PADD ARRAY
		//==================================================

		// work
		ei_new = tx;
		while(ei_new < d_common.in2_pad_cumv_elem){

			// figure out row/col location in padded array
			row = (ei_new+1) % d_common.in2_pad_cumv_rows - 1;												// (0-n) row
			col = (ei_new+1) / d_common.in2_pad_cumv_rows + 1 - 1;											// (0-n) column
			if((ei_new+1) % d_common.in2_pad_cumv_rows == 0){
				row = d_common.in2_pad_cumv_rows - 1;
				col = col-1;
			}

			// execution
			if(	row > (d_common.in2_pad_add_rows-1) &&														// do if has numbers in original array
				row < (d_common.in2_pad_add_rows+d_common.in2_rows) && 
				col > (d_common.in2_pad_add_cols-1) && 
				col < (d_common.in2_pad_add_cols+d_common.in2_cols)){
				ori_row = row - d_common.in2_pad_add_rows;
				ori_col = col - d_common.in2_pad_add_cols;
				d_unique[bx].d_in2_pad_cumv[ei_new] = d_unique[bx].d_in2[ori_col*d_common.in2_rows+ori_row];
			}
			else{																			// do if otherwise
				d_unique[bx].d_in2_pad_cumv[ei_new] = 0;
			}

			// go for second round
			ei_new = ei_new + NUMBER_THREADS;

		}

		//==================================================
		//	SYNCHRONIZE THREADS
		//==================================================

		__syncthreads();

		//==================================================
		//	VERTICAL CUMULATIVE SUM
		//==================================================

		//work
		ei_new = tx;
		while(ei_new < d_common.in2_pad_cumv_cols){

			// figure out column position
			pos_ori = ei_new*d_common.in2_pad_cumv_rows;

			// variables
			sum = 0;
			
			// loop through all rows
			for(position = pos_ori; position < pos_ori+d_common.in2_pad_cumv_rows; position = position + 1){
				d_unique[bx].d_in2_pad_cumv[position] = d_unique[bx].d_in2_pad_cumv[position] + sum;
				sum = d_unique[bx].d_in2_pad_cumv[position];
			}

			// go for second round
			ei_new = ei_new + NUMBER_THREADS;

		}

		//====================================================================================================
		//	SYNCHRONIZE THREADS
		//====================================================================================================

		__syncthreads();

		//====================================================================================================
		//	SELECTION
		//====================================================================================================

		// work
		ei_new = tx;
		while(ei_new < d_common.in2_pad_cumv_sel_elem){

			// figure out row/col location in new matrix
			row = (ei_new+1) % d_common.in2_pad_cumv_sel_rows - 1;												// (0-n) row
			col = (ei_new+1) / d_common.in2_pad_cumv_sel_rows + 1 - 1;											// (0-n) column
			if((ei_new+1) % d_common.in2_pad_cumv_sel_rows == 0){
				row = d_common.in2_pad_cumv_sel_rows - 1;
				col = col-1;
			}

			// figure out corresponding location in old matrix and copy values to new matrix
			ori_row = row + d_common.in2_pad_cumv_sel_rowlow - 1;
			ori_col = col + d_common.in2_pad_cumv_sel_collow - 1;
			d_unique[bx].d_in2_pad_cumv_sel[ei_new] = d_unique[bx].d_in2_pad_cumv[ori_col*d_common.in2_pad_cumv_rows+ori_row];

			// go for second round
			ei_new = ei_new + NUMBER_THREADS;

		}

		//====================================================================================================
		//	SYNCHRONIZE THREADS
		//====================================================================================================

		__syncthreads();

		//====================================================================================================
		//	SELECTION 2, SUBTRACTION, HORIZONTAL CUMULATIVE SUM
		//====================================================================================================

		//==================================================
		//	SELECTION 2
		//==================================================

		// work
		ei_new = tx;
		while(ei_new < d_common.in2_sub_cumh_elem){

			// figure out row/col location in new matrix
			row = (ei_new+1) % d_common.in2_sub_cumh_rows - 1;												// (0-n) row
			col = (ei_new+1) / d_common.in2_sub_cumh_rows + 1 - 1;											// (0-n) column
			if((ei_new+1) % d_common.in2_sub_cumh_rows == 0){
				row = d_common.in2_sub_cumh_rows - 1;
				col = col-1;
			}

			// figure out corresponding location in old matrix and copy values to new matrix
			ori_row = row + d_common.in2_pad_cumv_sel2_rowlow - 1;
			ori_col = col + d_common.in2_pad_cumv_sel2_collow - 1;
			d_unique[bx].d_in2_sub_cumh[ei_new] = d_unique[bx].d_in2_pad_cumv[ori_col*d_common.in2_pad_cumv_rows+ori_row];

			// go for second round
			ei_new = ei_new + NUMBER_THREADS;

		}

		//==================================================
		//	SYNCHRONIZE THREADS
		//==================================================

		__syncthreads();

		//==================================================
		//	SUBTRACTION
		//==================================================
		
		// work
		ei_new = tx;
		while(ei_new < d_common.in2_sub_cumh_elem){

			// subtract
			d_unique[bx].d_in2_sub_cumh[ei_new] = d_unique[bx].d_in2_pad_cumv_sel[ei_new] - d_unique[bx].d_in2_sub_cumh[ei_new];

			// go for second round
			ei_new = ei_new + NUMBER_THREADS;

		}

		//==================================================
		//	SYNCHRONIZE THREADS
		//==================================================

		__syncthreads();

		//==================================================
		//	HORIZONTAL CUMULATIVE SUM
		//==================================================

		// work
		ei_new = tx;
		while(ei_new < d_common.in2_sub_cumh_rows){

			// figure out row position
			pos_ori = ei_new;

			// variables
			sum = 0;

			// loop through all rows
			for(position = pos_ori; position < pos_ori+d_common.in2_sub_cumh_elem; position = position + d_common.in2_sub_cumh_rows){
				d_unique[bx].d_in2_sub_cumh[position] = d_unique[bx].d_in2_sub_cumh[position] + sum;
				sum = d_unique[bx].d_in2_sub_cumh[position];
			}

			// go for second round
			ei_new = ei_new + NUMBER_THREADS;

		}

		//====================================================================================================
		//	SYNCHRONIZE THREADS
		//====================================================================================================

		__syncthreads();

		//====================================================================================================
		//	SELECTION
		//====================================================================================================

		// work
		ei_new = tx;
		while(ei_new < d_common.in2_sub_cumh_sel_elem){

			// figure out row/col location in new matrix
			row = (ei_new+1) % d_common.in2_sub_cumh_sel_rows - 1;												// (0-n) row
			col = (ei_new+1) / d_common.in2_sub_cumh_sel_rows + 1 - 1;											// (0-n) column
			if((ei_new+1) % d_common.in2_sub_cumh_sel_rows == 0){
				row = d_common.in2_sub_cumh_sel_rows - 1;
				col = col - 1;
			}

			// figure out corresponding location in old matrix and copy values to new matrix
			ori_row = row + d_common.in2_sub_cumh_sel_rowlow - 1;
			ori_col = col + d_common.in2_sub_cumh_sel_collow - 1;
			d_unique[bx].d_in2_sub_cumh_sel[ei_new] = d_unique[bx].d_in2_sub_cumh[ori_col*d_common.in2_sub_cumh_rows+ori_row];

			// go for second round
			ei_new = ei_new + NUMBER_THREADS;

		}

		//====================================================================================================
		//	SYNCHRONIZE THREADS
		//====================================================================================================

		__syncthreads();

		//====================================================================================================
		//	SELECTION 2, SUBTRACTION
		//====================================================================================================

		//==================================================
		//	SELECTION 2
		//==================================================

		// work
		ei_new = tx;
		while(ei_new < d_common.in2_sub2_elem){

			// figure out row/col location in new matrix
			row = (ei_new+1) % d_common.in2_sub2_rows - 1;												// (0-n) row
			col = (ei_new+1) / d_common.in2_sub2_rows + 1 - 1;											// (0-n) column
			if((ei_new+1) % d_common.in2_sub2_rows == 0){
				row = d_common.in2_sub2_rows - 1;
				col = col-1;
			}

			// figure out corresponding location in old matrix and copy values to new matrix
			ori_row = row + d_common.in2_sub_cumh_sel2_rowlow - 1;
			ori_col = col + d_common.in2_sub_cumh_sel2_collow - 1;
			d_unique[bx].d_in2_sub2[ei_new] = d_unique[bx].d_in2_sub_cumh[ori_col*d_common.in2_sub_cumh_rows+ori_row];

			// go for second round
			ei_new = ei_new + NUMBER_THREADS;

		}

		//==================================================
		//	SYNCHRONIZE THREADS
		//==================================================

		__syncthreads();

		//==================================================
		//	SUBTRACTION
		//==================================================

		// work
		ei_new = tx;
		while(ei_new < d_common.in2_sub2_elem){

			// subtract
			d_unique[bx].d_in2_sub2[ei_new] = d_unique[bx].d_in2_sub_cumh_sel[ei_new] - d_unique[bx].d_in2_sub2[ei_new];

			// go for second round
			ei_new = ei_new + NUMBER_THREADS;

		}

		//======================================================================================================================================================
		//	SYNCHRONIZE THREADS
		//======================================================================================================================================================

		__syncthreads();

		//======================================================================================================================================================
		//	CUMULATIVE SUM 2
		//======================================================================================================================================================

		//====================================================================================================
		//	MULTIPLICATION
		//====================================================================================================

		// work
		ei_new = tx;
		while(ei_new < d_common.in2_sqr_elem){

			temp = d_unique[bx].d_in2[ei_new];
			d_unique[bx].d_in2_sqr[ei_new] = temp * temp;

			// go for second round
			ei_new = ei_new + NUMBER_THREADS;

		}

		//====================================================================================================
		//	SYNCHRONIZE THREADS
		//====================================================================================================

		__syncthreads();

		//====================================================================================================
		//	PAD ARRAY, VERTICAL CUMULATIVE SUM
		//====================================================================================================

		//==================================================
		//	PAD ARRAY
		//==================================================

		// work
		ei_new = tx;
		while(ei_new < d_common.in2_pad_cumv_elem){

			// figure out row/col location in padded array
			row = (ei_new+1) % d_common.in2_pad_cumv_rows - 1;												// (0-n) row
			col = (ei_new+1) / d_common.in2_pad_cumv_rows + 1 - 1;											// (0-n) column
			if((ei_new+1) % d_common.in2_pad_cumv_rows == 0){
				row = d_common.in2_pad_cumv_rows - 1;
				col = col-1;
			}

			// execution
			if(	row > (d_common.in2_pad_add_rows-1) &&													// do if has numbers in original array
				row < (d_common.in2_pad_add_rows+d_common.in2_sqr_rows) && 
				col > (d_common.in2_pad_add_cols-1) && 
				col < (d_common.in2_pad_add_cols+d_common.in2_sqr_cols)){
				ori_row = row - d_common.in2_pad_add_rows;
				ori_col = col - d_common.in2_pad_add_cols;
				d_unique[bx].d_in2_pad_cumv[ei_new] = d_unique[bx].d_in2_sqr[ori_col*d_common.in2_sqr_rows+ori_row];
			}
			else{																							// do if otherwise
				d_unique[bx].d_in2_pad_cumv[ei_new] = 0;
			}

			// go for second round
			ei_new = ei_new + NUMBER_THREADS;

		}

		//==================================================
		//	SYNCHRONIZE THREADS
		//==================================================

		__syncthreads();

		//==================================================
		//	VERTICAL CUMULATIVE SUM
		//==================================================

		//work
		ei_new = tx;
		while(ei_new < d_common.in2_pad_cumv_cols){

			// figure out column position
			pos_ori = ei_new*d_common.in2_pad_cumv_rows;

			// variables
			sum = 0;
			
			// loop through all rows
			for(position = pos_ori; position < pos_ori+d_common.in2_pad_cumv_rows; position = position + 1){
				d_unique[bx].d_in2_pad_cumv[position] = d_unique[bx].d_in2_pad_cumv[position] + sum;
				sum = d_unique[bx].d_in2_pad_cumv[position];
			}

			// go for second round
			ei_new = ei_new + NUMBER_THREADS;

		}

		//====================================================================================================
		//	SYNCHRONIZE THREADS
		//====================================================================================================

		__syncthreads();

		//====================================================================================================
		//	SELECTION
		//====================================================================================================

		// work
		ei_new = tx;
		while(ei_new < d_common.in2_pad_cumv_sel_elem){

			// figure out row/col location in new matrix
			row = (ei_new+1) % d_common.in2_pad_cumv_sel_rows - 1;												// (0-n) row
			col = (ei_new+1) / d_common.in2_pad_cumv_sel_rows + 1 - 1;											// (0-n) column
			if((ei_new+1) % d_common.in2_pad_cumv_sel_rows == 0){
				row = d_common.in2_pad_cumv_sel_rows - 1;
				col = col-1;
			}

			// figure out corresponding location in old matrix and copy values to new matrix
			ori_row = row + d_common.in2_pad_cumv_sel_rowlow - 1;
			ori_col = col + d_common.in2_pad_cumv_sel_collow - 1;
			d_unique[bx].d_in2_pad_cumv_sel[ei_new] = d_unique[bx].d_in2_pad_cumv[ori_col*d_common.in2_pad_cumv_rows+ori_row];

			// go for second round
			ei_new = ei_new + NUMBER_THREADS;

		}

		//====================================================================================================
		//	SYNCHRONIZE THREADS
		//====================================================================================================

		__syncthreads();

		//====================================================================================================
		//	SELECTION 2, SUBTRACTION, HORIZONTAL CUMULATIVE SUM
		//====================================================================================================

		//==================================================
		//	SELECTION 2
		//==================================================

		// work
		ei_new = tx;
		while(ei_new < d_common.in2_sub_cumh_elem){

			// figure out row/col location in new matrix
			row = (ei_new+1) % d_common.in2_sub_cumh_rows - 1;												// (0-n) row
			col = (ei_new+1) / d_common.in2_sub_cumh_rows + 1 - 1;											// (0-n) column
			if((ei_new+1) % d_common.in2_sub_cumh_rows == 0){
				row = d_common.in2_sub_cumh_rows - 1;
				col = col-1;
			}

			// figure out corresponding location in old matrix and copy values to new matrix
			ori_row = row + d_common.in2_pad_cumv_sel2_rowlow - 1;
			ori_col = col + d_common.in2_pad_cumv_sel2_collow - 1;
			d_unique[bx].d_in2_sub_cumh[ei_new] = d_unique[bx].d_in2_pad_cumv[ori_col*d_common.in2_pad_cumv_rows+ori_row];

			// go for second round
			ei_new = ei_new + NUMBER_THREADS;

		}

		//==================================================
		//	SYNCHRONIZE THREADS
		//==================================================

		__syncthreads();

		//==================================================
		//	SUBTRACTION
		//==================================================

		// work
		ei_new = tx;
		while(ei_new < d_common.in2_sub_cumh_elem){

			// subtract
			d_unique[bx].d_in2_sub_cumh[ei_new] = d_unique[bx].d_in2_pad_cumv_sel[ei_new] - d_unique[bx].d_in2_sub_cumh[ei_new];

			// go for second round
			ei_new = ei_new + NUMBER_THREADS;

		}

		//==================================================
		//	HORIZONTAL CUMULATIVE SUM
		//==================================================

		// work
		ei_new = tx;
		while(ei_new < d_common.in2_sub_cumh_rows){

			// figure out row position
			pos_ori = ei_new;

			// variables
			sum = 0;

			// loop through all rows
			for(position = pos_ori; position < pos_ori+d_common.in2_sub_cumh_elem; position = position + d_common.in2_sub_cumh_rows){
				d_unique[bx].d_in2_sub_cumh[position] = d_unique[bx].d_in2_sub_cumh[position] + sum;
				sum = d_unique[bx].d_in2_sub_cumh[position];
			}

			// go for second round
			ei_new = ei_new + NUMBER_THREADS;

		}

		//====================================================================================================
		//	SYNCHRONIZE THREADS
		//====================================================================================================

		__syncthreads();

		//====================================================================================================
		//	SELECTION
		//====================================================================================================

		// work
		ei_new = tx;
		while(ei_new < d_common.in2_sub_cumh_sel_elem){

			// figure out row/col location in new matrix
			row = (ei_new+1) % d_common.in2_sub_cumh_sel_rows - 1;												// (0-n) row
			col = (ei_new+1) / d_common.in2_sub_cumh_sel_rows + 1 - 1;											// (0-n) column
			if((ei_new+1) % d_common.in2_sub_cumh_sel_rows == 0){
				row = d_common.in2_sub_cumh_sel_rows - 1;
				col = col - 1;
			}

			// figure out corresponding location in old matrix and copy values to new matrix
			ori_row = row + d_common.in2_sub_cumh_sel_rowlow - 1;
			ori_col = col + d_common.in2_sub_cumh_sel_collow - 1;
			d_unique[bx].d_in2_sub_cumh_sel[ei_new] = d_unique[bx].d_in2_sub_cumh[ori_col*d_common.in2_sub_cumh_rows+ori_row];

			// go for second round
			ei_new = ei_new + NUMBER_THREADS;

		}

		//====================================================================================================
		//	SYNCHRONIZE THREADS
		//====================================================================================================

		__syncthreads();

		//====================================================================================================
		//	SELECTION 2, SUBTRACTION
		//====================================================================================================

		//==================================================
		//	SELECTION 2
		//==================================================

		// work
		ei_new = tx;
		while(ei_new < d_common.in2_sub2_elem){

			// figure out row/col location in new matrix
			row = (ei_new+1) % d_common.in2_sub2_rows - 1;												// (0-n) row
			col = (ei_new+1) / d_common.in2_sub2_rows + 1 - 1;											// (0-n) column
			if((ei_new+1) % d_common.in2_sub2_rows == 0){
				row = d_common.in2_sub2_rows - 1;
				col = col-1;
			}

			// figure out corresponding location in old matrix and copy values to new matrix
			ori_row = row + d_common.in2_sub_cumh_sel2_rowlow - 1;
			ori_col = col + d_common.in2_sub_cumh_sel2_collow - 1;
			d_unique[bx].d_in2_sqr_sub2[ei_new] = d_unique[bx].d_in2_sub_cumh[ori_col*d_common.in2_sub_cumh_rows+ori_row];

			// go for second round
			ei_new = ei_new + NUMBER_THREADS;

		}

		//==================================================
		//	SYNCHRONIZE THREADS
		//==================================================

		__syncthreads();

		//==================================================
		//	SUBTRACTION
		//==================================================

		// work
		ei_new = tx;
		while(ei_new < d_common.in2_sub2_elem){

			// subtract
			d_unique[bx].d_in2_sqr_sub2[ei_new] = d_unique[bx].d_in2_sub_cumh_sel[ei_new] - d_unique[bx].d_in2_sqr_sub2[ei_new];

			// go for second round
			ei_new = ei_new + NUMBER_THREADS;

		}

		//======================================================================================================================================================
		//	SYNCHRONIZE THREADS
		//======================================================================================================================================================

		__syncthreads();

		//======================================================================================================================================================
		//	FINAL
		//======================================================================================================================================================

		//====================================================================================================
		//	DENOMINATOR A		SAVE RESULT IN CUMULATIVE SUM A2
		//====================================================================================================

		// work
		ei_new = tx;
		while(ei_new < d_common.in2_sub2_elem){

			temp = d_unique[bx].d_in2_sub2[ei_new];
			temp2 = d_unique[bx].d_in2_sqr_sub2[ei_new] - (temp * temp / d_common.in_elem);
			if(temp2 < 0){
				temp2 = 0;
			}
			d_unique[bx].d_in2_sqr_sub2[ei_new] = sqrt(temp2);
			

			// go for second round
			ei_new = ei_new + NUMBER_THREADS;

		}

		//====================================================================================================
		//	SYNCHRONIZE THREADS
		//====================================================================================================

		__syncthreads();

		//====================================================================================================
		//	MULTIPLICATION
		//====================================================================================================

		// work
		ei_new = tx;
		while(ei_new < d_common.in_sqr_elem){

			temp = d_in[ei_new];
			d_unique[bx].d_in_sqr[ei_new] = temp * temp;

			// go for second round
			ei_new = ei_new + NUMBER_THREADS;

		}

		//====================================================================================================
		//	SYNCHRONIZE THREADS
		//====================================================================================================

		__syncthreads();

		//====================================================================================================
		//	IN SUM
		//====================================================================================================

		// work
		ei_new = tx;
		while(ei_new < d_common.in_cols){

			sum = 0;
			for(i = 0; i < d_common.in_rows; i++){

				sum = sum + d_in[ei_new*d_common.in_rows+i];

			}
			in_partial_sum[ei_new] = sum;

			// go for second round
			ei_new = ei_new + NUMBER_THREADS;

		}

		//====================================================================================================
		//	SYNCHRONIZE THREADS
		//====================================================================================================

		__syncthreads();

		//====================================================================================================
		//	IN_SQR SUM
		//====================================================================================================

		ei_new = tx;
		while(ei_new < d_common.in_sqr_rows){
				
			sum = 0;
			for(i = 0; i < d_common.in_sqr_cols; i++){

				sum = sum + d_unique[bx].d_in_sqr[ei_new+d_common.in_sqr_rows*i];

			}
			in_sqr_partial_sum[ei_new] = sum;

			// go for second round
			ei_new = ei_new + NUMBER_THREADS;

		}

		//====================================================================================================
		//	SYNCHRONIZE THREADS
		//====================================================================================================

		__syncthreads();

		//====================================================================================================
		//	FINAL SUMMATION
		//====================================================================================================

		if(tx == 0){

			in_final_sum = 0;
			for(i = 0; i<d_common.in_cols; i++){
				in_final_sum = in_final_sum + in_partial_sum[i];
			}

		}else if(tx == 1){

			in_sqr_final_sum = 0;
			for(i = 0; i<d_common.in_sqr_cols; i++){
				in_sqr_final_sum = in_sqr_final_sum + in_sqr_partial_sum[i];
			}

		}

		//====================================================================================================
		//	SYNCHRONIZE THREADS
		//====================================================================================================

		__syncthreads();

		//====================================================================================================
		//	DENOMINATOR T
		//====================================================================================================

		if(tx == 0){

			mean = in_final_sum / d_common.in_elem;													// gets mean (average) value of element in ROI
			mean_sqr = mean * mean;
			variance  = (in_sqr_final_sum / d_common.in_elem) - mean_sqr;							// gets variance of ROI
			deviation = sqrt(variance);																// gets standard deviation of ROI

			denomT = sqrt(float(d_common.in_elem-1))*deviation;

		}

		//====================================================================================================
		//	SYNCHRONIZE THREADS
		//====================================================================================================

		__syncthreads();

		//====================================================================================================
		//	DENOMINATOR		SAVE RESULT IN CUMULATIVE SUM A2
		//====================================================================================================

		// work
		ei_new = tx;
		while(ei_new < d_common.in2_sub2_elem){

			d_unique[bx].d_in2_sqr_sub2[ei_new] = d_unique[bx].d_in2_sqr_sub2[ei_new] * denomT;

			// go for second round
			ei_new = ei_new + NUMBER_THREADS;

		}

		//====================================================================================================
		//	SYNCHRONIZE THREADS
		//====================================================================================================

		__syncthreads();

		//====================================================================================================
		//	NUMERATOR	SAVE RESULT IN CONVOLUTION
		//====================================================================================================

		// work
		ei_new = tx;
		while(ei_new < d_common.conv_elem){

			d_unique[bx].d_conv[ei_new] = d_unique[bx].d_conv[ei_new] - d_unique[bx].d_in2_sub2[ei_new] * in_final_sum / d_common.in_elem;

			// go for second round
			ei_new = ei_new + NUMBER_THREADS;

		}

		//====================================================================================================
		//	SYNCHRONIZE THREADS
		//====================================================================================================

		__syncthreads();

		//====================================================================================================
		//	CORRELATION	SAVE RESULT IN CUMULATIVE SUM A2
		//====================================================================================================

		// work
		ei_new = tx;
		while(ei_new < d_common.in2_sub2_elem){

			d_unique[bx].d_in2_sqr_sub2[ei_new] = d_unique[bx].d_conv[ei_new] / d_unique[bx].d_in2_sqr_sub2[ei_new];

			// go for second round
			ei_new = ei_new + NUMBER_THREADS;

		}

		//======================================================================================================================================================
		//	SYNCHRONIZE THREADS
		//======================================================================================================================================================

		__syncthreads();

		//======================================================================================================================================================
		//	TEMPLATE MASK CREATE
		//======================================================================================================================================================

		cent = d_common.sSize + d_common.tSize + 1;
		if(d_common_change.frame_no == 0){
			tMask_row = cent + d_unique[bx].d_Row[d_unique[bx].point_no] - d_unique[bx].d_Row[d_unique[bx].point_no] - 1;
			tMask_col = cent + d_unique[bx].d_Col[d_unique[bx].point_no] - d_unique[bx].d_Col[d_unique[bx].point_no] - 1;
		}
		else{
			pointer = d_common_change.frame_no-1+d_unique[bx].point_no*d_common.no_frames;
			tMask_row = cent + d_unique[bx].d_tRowLoc[pointer] - d_unique[bx].d_Row[d_unique[bx].point_no] - 1;
			tMask_col = cent + d_unique[bx].d_tColLoc[pointer] - d_unique[bx].d_Col[d_unique[bx].point_no] - 1;
		}


		//work
		ei_new = tx;
		while(ei_new < d_common.tMask_elem){

			location = tMask_col*d_common.tMask_rows + tMask_row;

			if(ei_new==location){
				d_unique[bx].d_tMask[ei_new] = 1;
			}
			else{
				d_unique[bx].d_tMask[ei_new] = 0;
			}

			//go for second round
			ei_new = ei_new + NUMBER_THREADS;

		}

		//======================================================================================================================================================
		//	SYNCHRONIZE THREADS
		//======================================================================================================================================================

		__syncthreads();

		//======================================================================================================================================================
		//	MASK CONVOLUTION
		//======================================================================================================================================================

		// work
		ei_new = tx;
		while(ei_new < d_common.mask_conv_elem){

			// figure out row/col location in array
			ic = (ei_new+1) % d_common.mask_conv_rows;												// (1-n)
			jc = (ei_new+1) / d_common.mask_conv_rows + 1;											// (1-n)
			if((ei_new+1) % d_common.mask_conv_rows == 0){
				ic = d_common.mask_conv_rows;
				jc = jc-1;
			}

			//
			j = jc + d_common.mask_conv_joffset;
			jp1 = j + 1;
			if(d_common.mask_cols < jp1){
				ja1 = jp1 - d_common.mask_cols;
			}
			else{
				ja1 = 1;
			}
			if(d_common.tMask_cols < j){
				ja2 = d_common.tMask_cols;
			}
			else{
				ja2 = j;
			}

			i = ic + d_common.mask_conv_ioffset;
			ip1 = i + 1;
			
			if(d_common.mask_rows < ip1){
				ia1 = ip1 - d_common.mask_rows;
			}
			else{
				ia1 = 1;
			}
			if(d_common.tMask_rows < i){
				ia2 = d_common.tMask_rows;
			}
			else{
				ia2 = i;
			}

			s = 0;

			for(ja=ja1; ja<=ja2; ja++){
				jb = jp1 - ja;
				for(ia=ia1; ia<=ia2; ia++){
					ib = ip1 - ia;
					s = s + d_unique[bx].d_tMask[d_common.tMask_rows*(ja-1)+ia-1] * 1;
				}
			}

			// //d_unique[bx].d_mask_conv[d_common.mask_conv_rows*(jc-1)+ic-1] = s;
			d_unique[bx].d_mask_conv[ei_new] = d_unique[bx].d_in2_sqr_sub2[ei_new] * s;

			// go for second round
			ei_new = ei_new + NUMBER_THREADS;

		}

		//======================================================================================================================================================
		//	SYNCHRONIZE THREADS
		//======================================================================================================================================================

		__syncthreads();

		//======================================================================================================================================================
		//	MAXIMUM VALUE
		//======================================================================================================================================================

		//====================================================================================================
		//	INITIAL SEARCH
		//====================================================================================================

		ei_new = tx;
		while(ei_new < d_common.mask_conv_rows){

			for(i=0; i<d_common.mask_conv_cols; i++){
				largest_coordinate_current = ei_new*d_common.mask_conv_rows+i;
				largest_value_current = abs(d_unique[bx].d_mask_conv[largest_coordinate_current]);
				if(largest_value_current > largest_value){
					largest_coordinate = largest_coordinate_current;
					largest_value = largest_value_current;
				}
			}
			par_max_coo[ei_new] = largest_coordinate;
			par_max_val[ei_new] = largest_value;

			// go for second round
			ei_new = ei_new + NUMBER_THREADS;

		}

		//====================================================================================================
		//	SYNCHRONIZE THREADS
		//====================================================================================================

		__syncthreads();

		//====================================================================================================
		//	FINAL SEARCH
		//====================================================================================================

		if(tx == 0){

			for(i = 0; i < d_common.mask_conv_rows; i++){
				if(par_max_val[i] > fin_max_val){
					fin_max_val = par_max_val[i];
					fin_max_coo = par_max_coo[i];
				}
			}

			// convert coordinate to row/col form
			largest_row = (fin_max_coo+1) % d_common.mask_conv_rows - 1;											// (0-n) row
			largest_col = (fin_max_coo+1) / d_common.mask_conv_rows;												// (0-n) column
			if((fin_max_coo+1) % d_common.mask_conv_rows == 0){
				largest_row = d_common.mask_conv_rows - 1;
				largest_col = largest_col - 1;
			}

			// calculate offset
			largest_row = largest_row + 1;																	// compensate to match MATLAB format (1-n)
			largest_col = largest_col + 1;																	// compensate to match MATLAB format (1-n)
			offset_row = largest_row - d_common.in_rows - (d_common.sSize - d_common.tSize);
			offset_col = largest_col - d_common.in_cols - (d_common.sSize - d_common.tSize);
			pointer = d_common_change.frame_no+d_unique[bx].point_no*d_common.no_frames;
			d_unique[bx].d_tRowLoc[pointer] = d_unique[bx].d_Row[d_unique[bx].point_no] + offset_row;
			d_unique[bx].d_tColLoc[pointer] = d_unique[bx].d_Col[d_unique[bx].point_no] + offset_col;

		}

		//======================================================================================================================================================
		//	SYNCHRONIZE THREADS
		//======================================================================================================================================================

		__syncthreads();

	}
	
	//===============================================================================================================================================================================================================
	//===============================================================================================================================================================================================================
	//	COORDINATE AND TEMPLATE UPDATE
	//===============================================================================================================================================================================================================
	//===============================================================================================================================================================================================================

	// time19 = clock();

	// if the last frame in the bath, update template
	if(d_common_change.frame_no != 0 && (d_common_change.frame_no)%10 == 0){

		// update coordinate
		loc_pointer = d_unique[bx].point_no*d_common.no_frames+d_common_change.frame_no;
		d_unique[bx].d_Row[d_unique[bx].point_no] = d_unique[bx].d_tRowLoc[loc_pointer];
		d_unique[bx].d_Col[d_unique[bx].point_no] = d_unique[bx].d_tColLoc[loc_pointer];

		// work
		ei_new = tx;
		while(ei_new < d_common.in_elem){

			// figure out row/col location in new matrix
			row = (ei_new+1) % d_common.in_rows - 1;												// (0-n) row
			col = (ei_new+1) / d_common.in_rows + 1 - 1;											// (0-n) column
			if((ei_new+1) % d_common.in_rows == 0){
				row = d_common.in_rows - 1;
				col = col-1;
			}

			// figure out row/col location in corresponding new template area in image and give to every thread (get top left corner and progress down and right)
			ori_row = d_unique[bx].d_Row[d_unique[bx].point_no] - 25 + row - 1;
			ori_col = d_unique[bx].d_Col[d_unique[bx].point_no] - 25 + col - 1;
			ori_pointer = ori_col*d_common.frame_rows+ori_row;

			// update template
			d_in[ei_new] = d_common.alpha*d_in[ei_new] + (1.00-d_common.alpha)*d_common_change.d_frame[ori_pointer];

			// go for second round
			ei_new = ei_new + NUMBER_THREADS;

		}

	}

}

	//===============================================================================================================================================================================================================
	//===============================================================================================================================================================================================================
	//	END OF FUNCTION
	//===============================================================================================================================================================================================================
	//===============================================================================================================================================================================================================

int heartmain(int argc, char ** argv) {
	//hipDeviceSetCacheConfig(hipFuncCachePreferL1);
	//======================================================================================================================================================
	//	VARIABLES
	//======================================================================================================================================================

	// CUDA kernel execution parameters
	dim3 threads;
	dim3 blocks;

	// counter
	int i;
	int frames_processed;

	// frames
	char* video_file_name;
	avi_t* frames;
	fp* frame;

	//======================================================================================================================================================
	// 	FRAME
	//======================================================================================================================================================

	if(argc!=3){
		printf("ERROR: usage: heartwall <inputfile> <num of frames>\n");
		exit(1);
	}
	
	// open movie file
 	video_file_name = argv[1];
	frames = (avi_t*)AVI_open_input_file(video_file_name, 1);														// added casting
	if (frames == NULL)  {
		   AVI_print_error((char *) "Error with AVI_open_input_file");
		   return -1;
	}

	// common
	common.no_frames = AVI_video_frames(frames);
	common.frame_rows = AVI_video_height(frames);
	common.frame_cols = AVI_video_width(frames);
	common.frame_elem = common.frame_rows * common.frame_cols;
	common.frame_mem = sizeof(fp) * common.frame_elem;

	// pointers
	hipMalloc((void **)&common_change.d_frame, common.frame_mem);

	//======================================================================================================================================================
	// 	CHECK INPUT ARGUMENTS
	//======================================================================================================================================================
	
	frames_processed = atoi(argv[2]);
		if(frames_processed<0 || frames_processed>common.no_frames){
			printf("ERROR: %d is an incorrect number of frames specified, select in the range of 0-%d\n", frames_processed, common.no_frames);
			return 0;
	}
	

	//======================================================================================================================================================
	//	HARDCODED INPUTS FROM MATLAB
	//======================================================================================================================================================

	//====================================================================================================
	//	CONSTANTS
	//====================================================================================================

	common.sSize = 40;
	common.tSize = 25;
	common.maxMove = 10;
	common.alpha = 0.87;

	//====================================================================================================
	//	ENDO POINTS
	//====================================================================================================

	common.endoPoints = ENDO_POINTS;
	common.endo_mem = sizeof(int) * common.endoPoints;

	common.endoRow = (int *)malloc(common.endo_mem);
	common.endoRow[ 0] = 369;
	common.endoRow[ 1] = 400;
	common.endoRow[ 2] = 429;
	common.endoRow[ 3] = 452;
	common.endoRow[ 4] = 476;
	common.endoRow[ 5] = 486;
	common.endoRow[ 6] = 479;
	common.endoRow[ 7] = 458;
	common.endoRow[ 8] = 433;
	common.endoRow[ 9] = 404;
	common.endoRow[10] = 374;
	common.endoRow[11] = 346;
	common.endoRow[12] = 318;
	common.endoRow[13] = 294;
	common.endoRow[14] = 277;
	common.endoRow[15] = 269;
	common.endoRow[16] = 275;
	common.endoRow[17] = 287;
	common.endoRow[18] = 311;
	common.endoRow[19] = 339;
	hipMalloc((void **)&common.d_endoRow, common.endo_mem);
	hipMemcpy(common.d_endoRow, common.endoRow, common.endo_mem, hipMemcpyHostToDevice);

	common.endoCol = (int *)malloc(common.endo_mem);
	common.endoCol[ 0] = 408;
	common.endoCol[ 1] = 406;
	common.endoCol[ 2] = 397;
	common.endoCol[ 3] = 383;
	common.endoCol[ 4] = 354;
	common.endoCol[ 5] = 322;
	common.endoCol[ 6] = 294;
	common.endoCol[ 7] = 270;
	common.endoCol[ 8] = 250;
	common.endoCol[ 9] = 237;
	common.endoCol[10] = 235;
	common.endoCol[11] = 241;
	common.endoCol[12] = 254;
	common.endoCol[13] = 273;
	common.endoCol[14] = 300;
	common.endoCol[15] = 328;
	common.endoCol[16] = 356;
	common.endoCol[17] = 383;
	common.endoCol[18] = 401;
	common.endoCol[19] = 411;
	hipMalloc((void **)&common.d_endoCol, common.endo_mem);
	hipMemcpy(common.d_endoCol, common.endoCol, common.endo_mem, hipMemcpyHostToDevice);

	common.tEndoRowLoc = (int *)malloc(common.endo_mem * common.no_frames);
	hipMalloc((void **)&common.d_tEndoRowLoc, common.endo_mem * common.no_frames);

	common.tEndoColLoc = (int *)malloc(common.endo_mem * common.no_frames);
	hipMalloc((void **)&common.d_tEndoColLoc, common.endo_mem * common.no_frames);

	//====================================================================================================
	//	EPI POINTS
	//====================================================================================================

	common.epiPoints = EPI_POINTS;
	common.epi_mem = sizeof(int) * common.epiPoints;

	common.epiRow = (int *)malloc(common.epi_mem);
	common.epiRow[ 0] = 390;
	common.epiRow[ 1] = 419;
	common.epiRow[ 2] = 448;
	common.epiRow[ 3] = 474;
	common.epiRow[ 4] = 501;
	common.epiRow[ 5] = 519;
	common.epiRow[ 6] = 535;
	common.epiRow[ 7] = 542;
	common.epiRow[ 8] = 543;
	common.epiRow[ 9] = 538;
	common.epiRow[10] = 528;
	common.epiRow[11] = 511;
	common.epiRow[12] = 491;
	common.epiRow[13] = 466;
	common.epiRow[14] = 438;
	common.epiRow[15] = 406;
	common.epiRow[16] = 376;
	common.epiRow[17] = 347;
	common.epiRow[18] = 318;
	common.epiRow[19] = 291;
	common.epiRow[20] = 275;
	common.epiRow[21] = 259;
	common.epiRow[22] = 256;
	common.epiRow[23] = 252;
	common.epiRow[24] = 252;
	common.epiRow[25] = 257;
	common.epiRow[26] = 266;
	common.epiRow[27] = 283;
	common.epiRow[28] = 305;
	common.epiRow[29] = 331;
	common.epiRow[30] = 360;
	hipMalloc((void **)&common.d_epiRow, common.epi_mem);
	hipMemcpy(common.d_epiRow, common.epiRow, common.epi_mem, hipMemcpyHostToDevice);

	common.epiCol = (int *)malloc(common.epi_mem);
	common.epiCol[ 0] = 457;
	common.epiCol[ 1] = 454;
	common.epiCol[ 2] = 446;
	common.epiCol[ 3] = 431;
	common.epiCol[ 4] = 411;
	common.epiCol[ 5] = 388;
	common.epiCol[ 6] = 361;
	common.epiCol[ 7] = 331;
	common.epiCol[ 8] = 301;
	common.epiCol[ 9] = 273;
	common.epiCol[10] = 243;
	common.epiCol[11] = 218;
	common.epiCol[12] = 196;
	common.epiCol[13] = 178;
	common.epiCol[14] = 166;
	common.epiCol[15] = 157;
	common.epiCol[16] = 155;
	common.epiCol[17] = 165;
	common.epiCol[18] = 177;
	common.epiCol[19] = 197;
	common.epiCol[20] = 218;
	common.epiCol[21] = 248;
	common.epiCol[22] = 276;
	common.epiCol[23] = 304;
	common.epiCol[24] = 333;
	common.epiCol[25] = 361;
	common.epiCol[26] = 391;
	common.epiCol[27] = 415;
	common.epiCol[28] = 434;
	common.epiCol[29] = 448;
	common.epiCol[30] = 455;
	hipMalloc((void **)&common.d_epiCol, common.epi_mem);
	hipMemcpy(common.d_epiCol, common.epiCol, common.epi_mem, hipMemcpyHostToDevice);

	common.tEpiRowLoc = (int *)malloc(common.epi_mem * common.no_frames);
	hipMalloc((void **)&common.d_tEpiRowLoc, common.epi_mem * common.no_frames);

	common.tEpiColLoc = (int *)malloc(common.epi_mem * common.no_frames);
	hipMalloc((void **)&common.d_tEpiColLoc, common.epi_mem * common.no_frames);

	//====================================================================================================
	//	ALL POINTS
	//====================================================================================================

	common.allPoints = ALL_POINTS;

	//======================================================================================================================================================
	// 	TEMPLATE SIZES
	//======================================================================================================================================================

	// common
	common.in_rows = common.tSize + 1 + common.tSize;
	common.in_cols = common.in_rows;
	common.in_elem = common.in_rows * common.in_cols;
	common.in_mem = sizeof(fp) * common.in_elem;

	//======================================================================================================================================================
	// 	CREATE ARRAY OF TEMPLATES FOR ALL POINTS
	//======================================================================================================================================================

	// common
	hipMalloc((void **)&common.d_endoT, common.in_mem * common.endoPoints);
	hipMalloc((void **)&common.d_epiT, common.in_mem * common.epiPoints);

	//======================================================================================================================================================
	//	SPECIFIC TO ENDO OR EPI TO BE SET HERE
	//======================================================================================================================================================

	for(i=0; i<common.endoPoints; i++){
		unique[i].point_no = i;
		unique[i].d_Row = common.d_endoRow;
		unique[i].d_Col = common.d_endoCol;
		unique[i].d_tRowLoc = common.d_tEndoRowLoc;
		unique[i].d_tColLoc = common.d_tEndoColLoc;
		unique[i].d_T = common.d_endoT;
	}
	for(i=common.endoPoints; i<common.allPoints; i++){
		unique[i].point_no = i-common.endoPoints;
		unique[i].d_Row = common.d_epiRow;
		unique[i].d_Col = common.d_epiCol;
		unique[i].d_tRowLoc = common.d_tEpiRowLoc;
		unique[i].d_tColLoc = common.d_tEpiColLoc;
		unique[i].d_T = common.d_epiT;
	}

	//======================================================================================================================================================
	// 	RIGHT TEMPLATE 	FROM 	TEMPLATE ARRAY
	//======================================================================================================================================================

	// pointers
	for(i=0; i<common.allPoints; i++){
		unique[i].in_pointer = unique[i].point_no * common.in_elem;
	}

	//======================================================================================================================================================
	// 	AREA AROUND POINT		FROM	FRAME
	//======================================================================================================================================================

	// common
	common.in2_rows = 2 * common.sSize + 1;
	common.in2_cols = 2 * common.sSize + 1;
	common.in2_elem = common.in2_rows * common.in2_cols;
	common.in2_mem = sizeof(float) * common.in2_elem;

	// pointers
	for(i=0; i<common.allPoints; i++){
		hipMalloc((void **)&unique[i].d_in2, common.in2_mem);
	}

	//======================================================================================================================================================
	// 	CONVOLUTION
	//======================================================================================================================================================

	// common
	common.conv_rows = common.in_rows + common.in2_rows - 1;												// number of rows in I
	common.conv_cols = common.in_cols + common.in2_cols - 1;												// number of columns in I
	common.conv_elem = common.conv_rows * common.conv_cols;												// number of elements
	common.conv_mem = sizeof(float) * common.conv_elem;
	common.ioffset = 0;
	common.joffset = 0;

	// pointers
	for(i=0; i<common.allPoints; i++){
		hipMalloc((void **)&unique[i].d_conv, common.conv_mem);
	}

	//======================================================================================================================================================
	// 	CUMULATIVE SUM
	//======================================================================================================================================================

	//====================================================================================================
	// 	PADDING OF ARRAY, VERTICAL CUMULATIVE SUM
	//====================================================================================================

	// common
	common.in2_pad_add_rows = common.in_rows;
	common.in2_pad_add_cols = common.in_cols;

	common.in2_pad_cumv_rows = common.in2_rows + 2*common.in2_pad_add_rows;
	common.in2_pad_cumv_cols = common.in2_cols + 2*common.in2_pad_add_cols;
	common.in2_pad_cumv_elem = common.in2_pad_cumv_rows * common.in2_pad_cumv_cols;
	common.in2_pad_cumv_mem = sizeof(float) * common.in2_pad_cumv_elem;

	// pointers
	for(i=0; i<common.allPoints; i++){
		hipMalloc((void **)&unique[i].d_in2_pad_cumv, common.in2_pad_cumv_mem);
	}

	//====================================================================================================
	// 	SELECTION
	//====================================================================================================

	// common
	common.in2_pad_cumv_sel_rowlow = 1 + common.in_rows;													// (1 to n+1)
	common.in2_pad_cumv_sel_rowhig = common.in2_pad_cumv_rows - 1;
	common.in2_pad_cumv_sel_collow = 1;
	common.in2_pad_cumv_sel_colhig = common.in2_pad_cumv_cols;
	common.in2_pad_cumv_sel_rows = common.in2_pad_cumv_sel_rowhig - common.in2_pad_cumv_sel_rowlow + 1;
	common.in2_pad_cumv_sel_cols = common.in2_pad_cumv_sel_colhig - common.in2_pad_cumv_sel_collow + 1;
	common.in2_pad_cumv_sel_elem = common.in2_pad_cumv_sel_rows * common.in2_pad_cumv_sel_cols;
	common.in2_pad_cumv_sel_mem = sizeof(float) * common.in2_pad_cumv_sel_elem;

	// pointers
	for(i=0; i<common.allPoints; i++){
		hipMalloc((void **)&unique[i].d_in2_pad_cumv_sel, common.in2_pad_cumv_sel_mem);
	}

	//====================================================================================================
	// 	SELECTION	2, SUBTRACTION, HORIZONTAL CUMULATIVE SUM
	//====================================================================================================

	// common
	common.in2_pad_cumv_sel2_rowlow = 1;
	common.in2_pad_cumv_sel2_rowhig = common.in2_pad_cumv_rows - common.in_rows - 1;
	common.in2_pad_cumv_sel2_collow = 1;
	common.in2_pad_cumv_sel2_colhig = common.in2_pad_cumv_cols;
	common.in2_sub_cumh_rows = common.in2_pad_cumv_sel2_rowhig - common.in2_pad_cumv_sel2_rowlow + 1;
	common.in2_sub_cumh_cols = common.in2_pad_cumv_sel2_colhig - common.in2_pad_cumv_sel2_collow + 1;
	common.in2_sub_cumh_elem = common.in2_sub_cumh_rows * common.in2_sub_cumh_cols;
	common.in2_sub_cumh_mem = sizeof(float) * common.in2_sub_cumh_elem;

	// pointers
	for(i=0; i<common.allPoints; i++){
		hipMalloc((void **)&unique[i].d_in2_sub_cumh, common.in2_sub_cumh_mem);
	}

	//====================================================================================================
	// 	SELECTION
	//====================================================================================================

	// common
	common.in2_sub_cumh_sel_rowlow = 1;
	common.in2_sub_cumh_sel_rowhig = common.in2_sub_cumh_rows;
	common.in2_sub_cumh_sel_collow = 1 + common.in_cols;
	common.in2_sub_cumh_sel_colhig = common.in2_sub_cumh_cols - 1;
	common.in2_sub_cumh_sel_rows = common.in2_sub_cumh_sel_rowhig - common.in2_sub_cumh_sel_rowlow + 1;
	common.in2_sub_cumh_sel_cols = common.in2_sub_cumh_sel_colhig - common.in2_sub_cumh_sel_collow + 1;
	common.in2_sub_cumh_sel_elem = common.in2_sub_cumh_sel_rows * common.in2_sub_cumh_sel_cols;
	common.in2_sub_cumh_sel_mem = sizeof(float) * common.in2_sub_cumh_sel_elem;

	// pointers
	for(i=0; i<common.allPoints; i++){
		hipMalloc((void **)&unique[i].d_in2_sub_cumh_sel, common.in2_sub_cumh_sel_mem);
	}

	//====================================================================================================
	//	SELECTION 2, SUBTRACTION
	//====================================================================================================

	// common
	common.in2_sub_cumh_sel2_rowlow = 1;
	common.in2_sub_cumh_sel2_rowhig = common.in2_sub_cumh_rows;
	common.in2_sub_cumh_sel2_collow = 1;
	common.in2_sub_cumh_sel2_colhig = common.in2_sub_cumh_cols - common.in_cols - 1;
	common.in2_sub2_rows = common.in2_sub_cumh_sel2_rowhig - common.in2_sub_cumh_sel2_rowlow + 1;
	common.in2_sub2_cols = common.in2_sub_cumh_sel2_colhig - common.in2_sub_cumh_sel2_collow + 1;
	common.in2_sub2_elem = common.in2_sub2_rows * common.in2_sub2_cols;
	common.in2_sub2_mem = sizeof(float) * common.in2_sub2_elem;

	// pointers
	for(i=0; i<common.allPoints; i++){
		hipMalloc((void **)&unique[i].d_in2_sub2, common.in2_sub2_mem);
	}

	//======================================================================================================================================================
	//	CUMULATIVE SUM 2
	//======================================================================================================================================================

	//====================================================================================================
	//	MULTIPLICATION
	//====================================================================================================

	// common
	common.in2_sqr_rows = common.in2_rows;
	common.in2_sqr_cols = common.in2_cols;
	common.in2_sqr_elem = common.in2_elem;
	common.in2_sqr_mem = common.in2_mem;

	// pointers
	for(i=0; i<common.allPoints; i++){
		hipMalloc((void **)&unique[i].d_in2_sqr, common.in2_sqr_mem);
	}

	//====================================================================================================
	//	SELECTION 2, SUBTRACTION
	//====================================================================================================

	// common
	common.in2_sqr_sub2_rows = common.in2_sub2_rows;
	common.in2_sqr_sub2_cols = common.in2_sub2_cols;
	common.in2_sqr_sub2_elem = common.in2_sub2_elem;
	common.in2_sqr_sub2_mem = common.in2_sub2_mem;

	// pointers
	for(i=0; i<common.allPoints; i++){
		hipMalloc((void **)&unique[i].d_in2_sqr_sub2, common.in2_sqr_sub2_mem);
	}

	//======================================================================================================================================================
	//	FINAL
	//======================================================================================================================================================

	// common
	common.in_sqr_rows = common.in_rows;
	common.in_sqr_cols = common.in_cols;
	common.in_sqr_elem = common.in_elem;
	common.in_sqr_mem = common.in_mem;

	// pointers
	for(i=0; i<common.allPoints; i++){
		hipMalloc((void **)&unique[i].d_in_sqr, common.in_sqr_mem);
	}

	//======================================================================================================================================================
	//	TEMPLATE MASK CREATE
	//======================================================================================================================================================

	// common
	common.tMask_rows = common.in_rows + (common.sSize+1+common.sSize) - 1;
	common.tMask_cols = common.tMask_rows;
	common.tMask_elem = common.tMask_rows * common.tMask_cols;
	common.tMask_mem = sizeof(float) * common.tMask_elem;

	// pointers
	for(i=0; i<common.allPoints; i++){
		hipMalloc((void **)&unique[i].d_tMask, common.tMask_mem);
	}

	//======================================================================================================================================================
	//	POINT MASK INITIALIZE
	//======================================================================================================================================================

	// common
	common.mask_rows = common.maxMove;
	common.mask_cols = common.mask_rows;
	common.mask_elem = common.mask_rows * common.mask_cols;
	common.mask_mem = sizeof(float) * common.mask_elem;

	//======================================================================================================================================================
	//	MASK CONVOLUTION
	//======================================================================================================================================================

	// common
	common.mask_conv_rows = common.tMask_rows;												// number of rows in I
	common.mask_conv_cols = common.tMask_cols;												// number of columns in I
	common.mask_conv_elem = common.mask_conv_rows * common.mask_conv_cols;												// number of elements
	common.mask_conv_mem = sizeof(float) * common.mask_conv_elem;
	common.mask_conv_ioffset = (common.mask_rows-1)/2;
	if((common.mask_rows-1) % 2 > 0.5){
		common.mask_conv_ioffset = common.mask_conv_ioffset + 1;
	}
	common.mask_conv_joffset = (common.mask_cols-1)/2;
	if((common.mask_cols-1) % 2 > 0.5){
		common.mask_conv_joffset = common.mask_conv_joffset + 1;
	}

	// pointers
	for(i=0; i<common.allPoints; i++){
		hipMalloc((void **)&unique[i].d_mask_conv, common.mask_conv_mem);
	}

	//======================================================================================================================================================
	//	KERNEL
	//======================================================================================================================================================

	//====================================================================================================
	//	THREAD BLOCK
	//====================================================================================================

	// All kernels operations within kernel use same max size of threads. Size of block size is set to the size appropriate for max size operation (on padded matrix). Other use subsets of that.
	threads.x = NUMBER_THREADS;											// define the number of threads in the block
	threads.y = 1;
	blocks.x = common.allPoints;							// define the number of blocks in the grid
	blocks.y = 1;

	//====================================================================================================
	//	COPY ARGUMENTS
	//====================================================================================================

	hipMemcpyToSymbol(HIP_SYMBOL(d_common), &common, sizeof(params_common));
	hipMemcpyToSymbol(HIP_SYMBOL(d_unique), &unique, sizeof(params_unique)*ALL_POINTS);

	//====================================================================================================
	//	PRINT FRAME PROGRESS START
	//====================================================================================================

	//printf("frame progress: ");
	//fflush(NULL);

	//====================================================================================================
	//	LAUNCH
	//====================================================================================================

    hipDeviceSynchronize();
    
    ///
	common_change.frame_no=0;
	frame = get_frame(	frames,						// pointer to video file
										common_change.frame_no,				// number of frame that needs to be returned
										0,								// cropped?
										0,								// scaled?
										1);							// converted
	hipMemcpy(common_change.d_frame, frame, common.frame_mem, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_common_change), &common_change, sizeof(params_common_change));
	
    ///
    
    /*long long start_time; // = get_time();
    long long end_time; // = get_time();
    double time_elapsed = 0.0; // = elapsed_time(start_time, end_time);

	
		printf("blocks: %d, %d, %d, threads: %d, %d, %d\n", blocks.x, blocks.y, blocks.z, threads.x, threads.y, threads.z);

	/*for(common_change.frame_no=0; common_change.frame_no<frames_processed; common_change.frame_no++){

		// Extract a cropped version of the first frame from the video file
		frame = get_frame(	frames,						// pointer to video file
										common_change.frame_no,				// number of frame that needs to be returned
										0,								// cropped?
										0,								// scaled?
										1);							// converted

		// copy frame to GPU memory
		hipMemcpy(common_change.d_frame, frame, common.frame_mem, hipMemcpyHostToDevice);
		hipMemcpyToSymbol(HIP_SYMBOL(d_common_change), &common_change, sizeof(params_common_change));




		hipDeviceSynchronize();
		start_time = get_time();

		// launch GPU kernel
		heartwall_kernel<<<blocks, threads>>>();

		hipDeviceSynchronize();
		
		end_time = get_time();

		hipError_t errmsg = hipGetLastError();

    if ( hipSuccess != errmsg ) 
		{
				printf("Error msg: %d, hipGetErrorString: %s\n", errmsg, hipGetErrorString(errmsg));
        printf( "Kernel Execution Error!\n" );
		}

		time_elapsed += elapsed_time(start_time, end_time);


		// free frame after each loop iteration, since AVI library allocates memory for every frame fetched
		free(frame);

		// print frame progress
		printf("%d (time %lf)", common_change.frame_no, time_elapsed);
		fflush(NULL);

	}

	//====================================================================================================
	//	PRINT FRAME PROGRESS END
	//====================================================================================================

	printf("\n");
  printf("HIRREG time: %lf\n", time_elapsed);
	fflush(NULL);*/
}

__device__ void cfdkernel(int nelr, int* elements_surrounding_elements, float* normals, float* variables, float* fluxes)
{
	const float smoothing_coefficient = float(0.2f);
	const int i = (blockDim.x*blockIdx.x + threadIdx.x);

	int j, nb;
	float3 normal; float normal_len;
	float factor;

	float density_i = variables[i + VAR_DENSITY*nelr];
	float3 momentum_i;
	momentum_i.x = variables[i + (VAR_MOMENTUM+0)*nelr];
	momentum_i.y = variables[i + (VAR_MOMENTUM+1)*nelr];
	momentum_i.z = variables[i + (VAR_MOMENTUM+2)*nelr];

	float density_energy_i = variables[i + VAR_DENSITY_ENERGY*nelr];

	float3 velocity_i;             				compute_velocity(density_i, momentum_i, velocity_i);
	float speed_sqd_i                          = compute_speed_sqd(velocity_i);
	float speed_i                              = sqrtf(speed_sqd_i);
	float pressure_i                           = compute_pressure(density_i, density_energy_i, speed_sqd_i);
	float speed_of_sound_i                     = compute_speed_of_sound(density_i, pressure_i);
	float3 flux_contribution_i_momentum_x, flux_contribution_i_momentum_y, flux_contribution_i_momentum_z;
	float3 flux_contribution_i_density_energy;	
	compute_flux_contribution(density_i, momentum_i, density_energy_i, pressure_i, velocity_i, flux_contribution_i_momentum_x, flux_contribution_i_momentum_y, flux_contribution_i_momentum_z, flux_contribution_i_density_energy);

	float flux_i_density = float(0.0f);
	float3 flux_i_momentum;
	flux_i_momentum.x = float(0.0f);
	flux_i_momentum.y = float(0.0f);
	flux_i_momentum.z = float(0.0f);
	float flux_i_density_energy = float(0.0f);
	
	float3 velocity_nb;
	float density_nb, density_energy_nb;
	float3 momentum_nb;
	float3 flux_contribution_nb_momentum_x, flux_contribution_nb_momentum_y, flux_contribution_nb_momentum_z;
	float3 flux_contribution_nb_density_energy;	
	float speed_sqd_nb, speed_of_sound_nb, pressure_nb;

	#pragma unroll
	for(j = 0; j < NNB; j++)
	{
		nb = elements_surrounding_elements[i + j*nelr];
		normal.x = normals[i + (j + 0*NNB)*nelr];
		normal.y = normals[i + (j + 1*NNB)*nelr];
		normal.z = normals[i + (j + 2*NNB)*nelr];
		normal_len = sqrtf(normal.x*normal.x + normal.y*normal.y + normal.z*normal.z);
	
		if(nb >= 0) 	// a legitimate neighbor
		{
			density_nb = variables[nb + VAR_DENSITY*nelr];
			momentum_nb.x = variables[nb + (VAR_MOMENTUM+0)*nelr];
			momentum_nb.y = variables[nb + (VAR_MOMENTUM+1)*nelr];
			momentum_nb.z = variables[nb + (VAR_MOMENTUM+2)*nelr];
			density_energy_nb = variables[nb + VAR_DENSITY_ENERGY*nelr];
												compute_velocity(density_nb, momentum_nb, velocity_nb);
			speed_sqd_nb                      = compute_speed_sqd(velocity_nb);
			pressure_nb                       = compute_pressure(density_nb, density_energy_nb, speed_sqd_nb);
			speed_of_sound_nb                 = compute_speed_of_sound(density_nb, pressure_nb);
				                            compute_flux_contribution(density_nb, momentum_nb, density_energy_nb, pressure_nb, velocity_nb, flux_contribution_nb_momentum_x, flux_contribution_nb_momentum_y, flux_contribution_nb_momentum_z, flux_contribution_nb_density_energy);
		
			// artificial viscosity
			factor = -normal_len*smoothing_coefficient*float(0.5f)*(speed_i + sqrtf(speed_sqd_nb) + speed_of_sound_i + speed_of_sound_nb);
			flux_i_density += factor*(density_i-density_nb);
			flux_i_density_energy += factor*(density_energy_i-density_energy_nb);
			flux_i_momentum.x += factor*(momentum_i.x-momentum_nb.x);
			flux_i_momentum.y += factor*(momentum_i.y-momentum_nb.y);
			flux_i_momentum.z += factor*(momentum_i.z-momentum_nb.z);

			// accumulate cell-centered fluxes
			factor = float(0.5f)*normal.x;
			flux_i_density += factor*(momentum_nb.x+momentum_i.x);
			flux_i_density_energy += factor*(flux_contribution_nb_density_energy.x+flux_contribution_i_density_energy.x);
			flux_i_momentum.x += factor*(flux_contribution_nb_momentum_x.x+flux_contribution_i_momentum_x.x);
			flux_i_momentum.y += factor*(flux_contribution_nb_momentum_y.x+flux_contribution_i_momentum_y.x);
			flux_i_momentum.z += factor*(flux_contribution_nb_momentum_z.x+flux_contribution_i_momentum_z.x);
		
			factor = float(0.5f)*normal.y;
			flux_i_density += factor*(momentum_nb.y+momentum_i.y);
			flux_i_density_energy += factor*(flux_contribution_nb_density_energy.y+flux_contribution_i_density_energy.y);
			flux_i_momentum.x += factor*(flux_contribution_nb_momentum_x.y+flux_contribution_i_momentum_x.y);
			flux_i_momentum.y += factor*(flux_contribution_nb_momentum_y.y+flux_contribution_i_momentum_y.y);
			flux_i_momentum.z += factor*(flux_contribution_nb_momentum_z.y+flux_contribution_i_momentum_z.y);
		
			factor = float(0.5f)*normal.z;
			flux_i_density += factor*(momentum_nb.z+momentum_i.z);
			flux_i_density_energy += factor*(flux_contribution_nb_density_energy.z+flux_contribution_i_density_energy.z);
			flux_i_momentum.x += factor*(flux_contribution_nb_momentum_x.z+flux_contribution_i_momentum_x.z);
			flux_i_momentum.y += factor*(flux_contribution_nb_momentum_y.z+flux_contribution_i_momentum_y.z);
			flux_i_momentum.z += factor*(flux_contribution_nb_momentum_z.z+flux_contribution_i_momentum_z.z);
		}
		else if(nb == -1)	// a wing boundary
		{
			flux_i_momentum.x += normal.x*pressure_i;
			flux_i_momentum.y += normal.y*pressure_i;
			flux_i_momentum.z += normal.z*pressure_i;
		}
		else if(nb == -2) // a far field boundary
		{
			factor = float(0.5f)*normal.x;
			flux_i_density += factor*(ff_variable[VAR_MOMENTUM+0]+momentum_i.x);
			flux_i_density_energy += factor*(ff_flux_contribution_density_energy[0].x+flux_contribution_i_density_energy.x);
			flux_i_momentum.x += factor*(ff_flux_contribution_momentum_x[0].x + flux_contribution_i_momentum_x.x);
			flux_i_momentum.y += factor*(ff_flux_contribution_momentum_y[0].x + flux_contribution_i_momentum_y.x);
			flux_i_momentum.z += factor*(ff_flux_contribution_momentum_z[0].x + flux_contribution_i_momentum_z.x);
		
			factor = float(0.5f)*normal.y;
			flux_i_density += factor*(ff_variable[VAR_MOMENTUM+1]+momentum_i.y);
			flux_i_density_energy += factor*(ff_flux_contribution_density_energy[0].y+flux_contribution_i_density_energy.y);
			flux_i_momentum.x += factor*(ff_flux_contribution_momentum_x[0].y + flux_contribution_i_momentum_x.y);
			flux_i_momentum.y += factor*(ff_flux_contribution_momentum_y[0].y + flux_contribution_i_momentum_y.y);
			flux_i_momentum.z += factor*(ff_flux_contribution_momentum_z[0].y + flux_contribution_i_momentum_z.y);

			factor = float(0.5f)*normal.z;
			flux_i_density += factor*(ff_variable[VAR_MOMENTUM+2]+momentum_i.z);
			flux_i_density_energy += factor*(ff_flux_contribution_density_energy[0].z+flux_contribution_i_density_energy.z);
			flux_i_momentum.x += factor*(ff_flux_contribution_momentum_x[0].z + flux_contribution_i_momentum_x.z);
			flux_i_momentum.y += factor*(ff_flux_contribution_momentum_y[0].z + flux_contribution_i_momentum_y.z);
			flux_i_momentum.z += factor*(ff_flux_contribution_momentum_z[0].z + flux_contribution_i_momentum_z.z);

		}
	}

	fluxes[i + VAR_DENSITY*nelr] = flux_i_density;
	fluxes[i + (VAR_MOMENTUM+0)*nelr] = flux_i_momentum.x;
	fluxes[i + (VAR_MOMENTUM+1)*nelr] = flux_i_momentum.y;
	fluxes[i + (VAR_MOMENTUM+2)*nelr] = flux_i_momentum.z;
	fluxes[i + VAR_DENSITY_ENERGY*nelr] = flux_i_density_energy;
}

__global__ void kernel(int nelr, int* elements_surrounding_elements, float* normals, float* variables, float* fluxes)
{
	if(threadIdx.x % 3 == 1) {
		const float smoothing_coefficient = float(0.2f);
		const int i = (blockDim.x*blockIdx.x + threadIdx.x);
	
		int j, nb;
		float3 normal; float normal_len;
		float factor;
	
		float density_i = variables[i + VAR_DENSITY*nelr];
		float3 momentum_i;
		momentum_i.x = variables[i + (VAR_MOMENTUM+0)*nelr];
		momentum_i.y = variables[i + (VAR_MOMENTUM+1)*nelr];
		momentum_i.z = variables[i + (VAR_MOMENTUM+2)*nelr];

		float density_energy_i = variables[i + VAR_DENSITY_ENERGY*nelr];

		float3 velocity_i;             				compute_velocity(density_i, momentum_i, velocity_i);
		float speed_sqd_i                          = compute_speed_sqd(velocity_i);
		float speed_i                              = sqrtf(speed_sqd_i);
		float pressure_i                           = compute_pressure(density_i, density_energy_i, speed_sqd_i);
		float speed_of_sound_i                     = compute_speed_of_sound(density_i, pressure_i);
		float3 flux_contribution_i_momentum_x, flux_contribution_i_momentum_y, flux_contribution_i_momentum_z;
		float3 flux_contribution_i_density_energy;	
		compute_flux_contribution(density_i, momentum_i, density_energy_i, pressure_i, velocity_i, flux_contribution_i_momentum_x, flux_contribution_i_momentum_y, flux_contribution_i_momentum_z, flux_contribution_i_density_energy);
	
		float flux_i_density = float(0.0f);
		float3 flux_i_momentum;
		flux_i_momentum.x = float(0.0f);
		flux_i_momentum.y = float(0.0f);
		flux_i_momentum.z = float(0.0f);
		float flux_i_density_energy = float(0.0f);
		
		float3 velocity_nb;
		float density_nb, density_energy_nb;
		float3 momentum_nb;
		float3 flux_contribution_nb_momentum_x, flux_contribution_nb_momentum_y, flux_contribution_nb_momentum_z;
		float3 flux_contribution_nb_density_energy;	
		float speed_sqd_nb, speed_of_sound_nb, pressure_nb;
	
		#pragma unroll
		for(j = 0; j < NNB; j++)
		{
			nb = elements_surrounding_elements[i + j*nelr];
			normal.x = normals[i + (j + 0*NNB)*nelr];
			normal.y = normals[i + (j + 1*NNB)*nelr];
			normal.z = normals[i + (j + 2*NNB)*nelr];
			normal_len = sqrtf(normal.x*normal.x + normal.y*normal.y + normal.z*normal.z);
		
			if(nb >= 0) 	// a legitimate neighbor
			{
				density_nb = variables[nb + VAR_DENSITY*nelr];
				momentum_nb.x = variables[nb + (VAR_MOMENTUM+0)*nelr];
				momentum_nb.y = variables[nb + (VAR_MOMENTUM+1)*nelr];
				momentum_nb.z = variables[nb + (VAR_MOMENTUM+2)*nelr];
				density_energy_nb = variables[nb + VAR_DENSITY_ENERGY*nelr];
													compute_velocity(density_nb, momentum_nb, velocity_nb);
				speed_sqd_nb                      = compute_speed_sqd(velocity_nb);
				pressure_nb                       = compute_pressure(density_nb, density_energy_nb, speed_sqd_nb);
				speed_of_sound_nb                 = compute_speed_of_sound(density_nb, pressure_nb);
					                            compute_flux_contribution(density_nb, momentum_nb, density_energy_nb, pressure_nb, velocity_nb, flux_contribution_nb_momentum_x, flux_contribution_nb_momentum_y, flux_contribution_nb_momentum_z, flux_contribution_nb_density_energy);
			
				// artificial viscosity
				factor = -normal_len*smoothing_coefficient*float(0.5f)*(speed_i + sqrtf(speed_sqd_nb) + speed_of_sound_i + speed_of_sound_nb);
				flux_i_density += factor*(density_i-density_nb);
				flux_i_density_energy += factor*(density_energy_i-density_energy_nb);
				flux_i_momentum.x += factor*(momentum_i.x-momentum_nb.x);
				flux_i_momentum.y += factor*(momentum_i.y-momentum_nb.y);
				flux_i_momentum.z += factor*(momentum_i.z-momentum_nb.z);

				// accumulate cell-centered fluxes
				factor = float(0.5f)*normal.x;
				flux_i_density += factor*(momentum_nb.x+momentum_i.x);
				flux_i_density_energy += factor*(flux_contribution_nb_density_energy.x+flux_contribution_i_density_energy.x);
				flux_i_momentum.x += factor*(flux_contribution_nb_momentum_x.x+flux_contribution_i_momentum_x.x);
				flux_i_momentum.y += factor*(flux_contribution_nb_momentum_y.x+flux_contribution_i_momentum_y.x);
				flux_i_momentum.z += factor*(flux_contribution_nb_momentum_z.x+flux_contribution_i_momentum_z.x);
			
				factor = float(0.5f)*normal.y;
				flux_i_density += factor*(momentum_nb.y+momentum_i.y);
				flux_i_density_energy += factor*(flux_contribution_nb_density_energy.y+flux_contribution_i_density_energy.y);
				flux_i_momentum.x += factor*(flux_contribution_nb_momentum_x.y+flux_contribution_i_momentum_x.y);
				flux_i_momentum.y += factor*(flux_contribution_nb_momentum_y.y+flux_contribution_i_momentum_y.y);
				flux_i_momentum.z += factor*(flux_contribution_nb_momentum_z.y+flux_contribution_i_momentum_z.y);
			
				factor = float(0.5f)*normal.z;
				flux_i_density += factor*(momentum_nb.z+momentum_i.z);
				flux_i_density_energy += factor*(flux_contribution_nb_density_energy.z+flux_contribution_i_density_energy.z);
				flux_i_momentum.x += factor*(flux_contribution_nb_momentum_x.z+flux_contribution_i_momentum_x.z);
				flux_i_momentum.y += factor*(flux_contribution_nb_momentum_y.z+flux_contribution_i_momentum_y.z);
				flux_i_momentum.z += factor*(flux_contribution_nb_momentum_z.z+flux_contribution_i_momentum_z.z);
			}
			else if(nb == -1)	// a wing boundary
			{
				flux_i_momentum.x += normal.x*pressure_i;
				flux_i_momentum.y += normal.y*pressure_i;
				flux_i_momentum.z += normal.z*pressure_i;
			}
			else if(nb == -2) // a far field boundary
			{
				factor = float(0.5f)*normal.x;
				flux_i_density += factor*(ff_variable[VAR_MOMENTUM+0]+momentum_i.x);
				flux_i_density_energy += factor*(ff_flux_contribution_density_energy[0].x+flux_contribution_i_density_energy.x);
				flux_i_momentum.x += factor*(ff_flux_contribution_momentum_x[0].x + flux_contribution_i_momentum_x.x);
				flux_i_momentum.y += factor*(ff_flux_contribution_momentum_y[0].x + flux_contribution_i_momentum_y.x);
				flux_i_momentum.z += factor*(ff_flux_contribution_momentum_z[0].x + flux_contribution_i_momentum_z.x);
			
				factor = float(0.5f)*normal.y;
				flux_i_density += factor*(ff_variable[VAR_MOMENTUM+1]+momentum_i.y);
				flux_i_density_energy += factor*(ff_flux_contribution_density_energy[0].y+flux_contribution_i_density_energy.y);
				flux_i_momentum.x += factor*(ff_flux_contribution_momentum_x[0].y + flux_contribution_i_momentum_x.y);
				flux_i_momentum.y += factor*(ff_flux_contribution_momentum_y[0].y + flux_contribution_i_momentum_y.y);
				flux_i_momentum.z += factor*(ff_flux_contribution_momentum_z[0].y + flux_contribution_i_momentum_z.y);

				factor = float(0.5f)*normal.z;
				flux_i_density += factor*(ff_variable[VAR_MOMENTUM+2]+momentum_i.z);
				flux_i_density_energy += factor*(ff_flux_contribution_density_energy[0].z+flux_contribution_i_density_energy.z);
				flux_i_momentum.x += factor*(ff_flux_contribution_momentum_x[0].z + flux_contribution_i_momentum_x.z);
				flux_i_momentum.y += factor*(ff_flux_contribution_momentum_y[0].z + flux_contribution_i_momentum_y.z);
				flux_i_momentum.z += factor*(ff_flux_contribution_momentum_z[0].z + flux_contribution_i_momentum_z.z);

			}
		}

		fluxes[i + VAR_DENSITY*nelr] = flux_i_density;
		fluxes[i + (VAR_MOMENTUM+0)*nelr] = flux_i_momentum.x;
		fluxes[i + (VAR_MOMENTUM+1)*nelr] = flux_i_momentum.y;
		fluxes[i + (VAR_MOMENTUM+2)*nelr] = flux_i_momentum.z;
		fluxes[i + VAR_DENSITY_ENERGY*nelr] = flux_i_density_energy;
	} else {
		heartwall_kernel();
	}
}

#define N 2

__global__ void Kernel1(int nelr, int* elements_surrounding_elements, float* normals, float* variables, float* fluxes)
{
    cfdkernel(nelr, elements_surrounding_elements, normals, variables, fluxes);
}

__global__ void Kernel2()
{
    heartwall_kernel();
}
void run_kernels(int nelr, int* elements_surrounding_elements, float* normals, float* variables, float* fluxes)
{
	dim3 Dg(nelr / block_length), Db(block_length);
	int numBlocks = nelr/block_length;
	if(numBlocks > ALL_POINTS){ 
		numBlocks = ALL_POINTS;
	}
    hipStream_t streams[N];
    for(int i = 0; i < N; i++)
        hipStreamCreate(&streams[i]);
	//kernel<<<numBlocks,Db>>>(nelr, elements_surrounding_elements, normals, variables, fluxes);
    Kernel1<<<numBlocks/3,Db, 0, streams[0]>>>(nelr, elements_surrounding_elements, normals, variables, fluxes);
    Kernel2<<<numBlocks-numBlocks/3,Db, 0, streams[1]>>>();
	getLastCudaError("kernel failed");
}

/*
 * Main function
 */
int main(int argc, char** argv)
{
	//hipDeviceSetCacheConfig(hipFuncCachePreferL1);
	
	if (argc < 2)
	{
		std::cout << "specify data file name" << std::endl;
		return 0;
	}
	const char* data_file_name = argv[1];
	
	hipDeviceProp_t prop;
	int dev;
	
	checkCudaErrors(hipSetDevice(0));
	checkCudaErrors(hipGetDevice(&dev));
	checkCudaErrors(hipGetDeviceProperties(&prop, dev));
	
	printf("Name:                     %s\n", prop.name);

	// set far field conditions and load them into constant memory on the gpu
	{
		float h_ff_variable[NVAR];
		const float angle_of_attack = float(3.1415926535897931 / 180.0f) * float(deg_angle_of_attack);
		
		h_ff_variable[VAR_DENSITY] = float(1.4);
		
		float ff_pressure = float(1.0f);
		float ff_speed_of_sound = sqrt(GAMMA*ff_pressure / h_ff_variable[VAR_DENSITY]);
		float ff_speed = float(ff_mach)*ff_speed_of_sound;
		
		float3 ff_velocity;
		ff_velocity.x = ff_speed*float(cos((float)angle_of_attack));
		ff_velocity.y = ff_speed*float(sin((float)angle_of_attack));
		ff_velocity.z = 0.0f;
		
		h_ff_variable[VAR_MOMENTUM+0] = h_ff_variable[VAR_DENSITY] * ff_velocity.x;
		h_ff_variable[VAR_MOMENTUM+1] = h_ff_variable[VAR_DENSITY] * ff_velocity.y;
		h_ff_variable[VAR_MOMENTUM+2] = h_ff_variable[VAR_DENSITY] * ff_velocity.z;
				
		h_ff_variable[VAR_DENSITY_ENERGY] = h_ff_variable[VAR_DENSITY]*(float(0.5f)*(ff_speed*ff_speed)) + (ff_pressure / float(GAMMA-1.0f));

		float3 h_ff_momentum;
		h_ff_momentum.x = *(h_ff_variable+VAR_MOMENTUM+0);
		h_ff_momentum.y = *(h_ff_variable+VAR_MOMENTUM+1);
		h_ff_momentum.z = *(h_ff_variable+VAR_MOMENTUM+2);
		float3 h_ff_flux_contribution_momentum_x;
		float3 h_ff_flux_contribution_momentum_y;
		float3 h_ff_flux_contribution_momentum_z;
		float3 h_ff_flux_contribution_density_energy;
		compute_flux_contribution(h_ff_variable[VAR_DENSITY], h_ff_momentum, h_ff_variable[VAR_DENSITY_ENERGY], ff_pressure, ff_velocity, h_ff_flux_contribution_momentum_x, h_ff_flux_contribution_momentum_y, h_ff_flux_contribution_momentum_z, h_ff_flux_contribution_density_energy);

		// copy far field conditions to the gpu
		checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(ff_variable),          h_ff_variable,          NVAR*sizeof(float)) );
		checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(ff_flux_contribution_momentum_x), &h_ff_flux_contribution_momentum_x, sizeof(float3)) );
		checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(ff_flux_contribution_momentum_y), &h_ff_flux_contribution_momentum_y, sizeof(float3)) );
		checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(ff_flux_contribution_momentum_z), &h_ff_flux_contribution_momentum_z, sizeof(float3)) );
		
		checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(ff_flux_contribution_density_energy), &h_ff_flux_contribution_density_energy, sizeof(float3)) );		
	}
	int nel;
	int nelr;
	
	// read in domain geometry
	float* areas;
	int* elements_surrounding_elements;
	float* normals;
	{
		std::ifstream file(data_file_name);
	
		file >> nel;
		nelr = block_length*((nel / block_length )+ std::min(1, nel % block_length));

		float* h_areas = new float[nelr];
		int* h_elements_surrounding_elements = new int[nelr*NNB];
		float* h_normals = new float[nelr*NDIM*NNB];

				
		// read in data
		for(int i = 0; i < nel; i++)
		{
			file >> h_areas[i];
			for(int j = 0; j < NNB; j++)
			{
				file >> h_elements_surrounding_elements[i + j*nelr];
				if(h_elements_surrounding_elements[i+j*nelr] < 0) h_elements_surrounding_elements[i+j*nelr] = -1;
				h_elements_surrounding_elements[i + j*nelr]--; //it's coming in with Fortran numbering				
				
				for(int k = 0; k < NDIM; k++)
				{
					file >> h_normals[i + (j + k*NNB)*nelr];
					h_normals[i + (j + k*NNB)*nelr] = -h_normals[i + (j + k*NNB)*nelr];
				}
			}
		}
		
		// fill in remaining data
		int last = nel-1;
		for(int i = nel; i < nelr; i++)
		{
			h_areas[i] = h_areas[last];
			for(int j = 0; j < NNB; j++)
			{
				// duplicate the last element
				h_elements_surrounding_elements[i + j*nelr] = h_elements_surrounding_elements[last + j*nelr];	
				for(int k = 0; k < NDIM; k++) h_normals[last + (j + k*NNB)*nelr] = h_normals[last + (j + k*NNB)*nelr];
			}
		}
		
		areas = alloc<float>(nelr);
		upload<float>(areas, h_areas, nelr);

		elements_surrounding_elements = alloc<int>(nelr*NNB);
		upload<int>(elements_surrounding_elements, h_elements_surrounding_elements, nelr*NNB);

		normals = alloc<float>(nelr*NDIM*NNB);
		upload<float>(normals, h_normals, nelr*NDIM*NNB);
				
		delete[] h_areas;
		delete[] h_elements_surrounding_elements;
		delete[] h_normals;
	}

	// Create arrays and set initial conditions
	float* variables = alloc<float>(nelr*NVAR);
	initialize_variables(nelr, variables);

	float* old_variables = alloc<float>(nelr*NVAR);   	
	float* fluxes = alloc<float>(nelr*NVAR);
	float* step_factors = alloc<float>(nelr); 

	// make sure all memory is floatly allocated before we start timing
	initialize_variables(nelr, old_variables);
	initialize_variables(nelr, fluxes);
	hipMemset( (void*) step_factors, 0, sizeof(float)*nelr );
	// make sure CUDA isn't still doing something before we start timing
	hipDeviceSynchronize();

	// these need to be computed the first time in order to compute time step
	std::cout << "Starting..." << std::endl;

	StopWatchInterface *timer = 0;
	  //	unsigned int timer = 0;
	
	
	heartmain(argc-1, argv+1);
	
	// CUT_SAFE_CALL( cutCreateTimer( &timer));
	// CUT_SAFE_CALL( cutStartTimer( timer));
	sdkCreateTimer(&timer); 
	sdkStartTimer(&timer);
	run_kernels(nelr, elements_surrounding_elements, normals, variables, fluxes);
	hipDeviceSynchronize();
	//	CUT_SAFE_CALL( cutStopTimer(timer) );  
	sdkStopTimer(&timer); 

	std::cout  << "runtime: " << sdkGetAverageTimerValue(&timer) << std::endl;

	
	std::cout << "Cleaning up..." << std::endl;
	dealloc<float>(areas);
	dealloc<int>(elements_surrounding_elements);
	dealloc<float>(normals);
	
	dealloc<float>(variables);
	dealloc<float>(old_variables);
	dealloc<float>(fluxes);
	dealloc<float>(step_factors);

	std::cout << "Done..." << std::endl;

	return 0;
}
