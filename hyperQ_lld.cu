#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define N 2

__global__ void Kernel1()
{
    if(threadIdx.x == 0 && blockIdx.x == 0)
        printf("Hello from 1\n");
}

__global__ void Kernel2()
{
    if(threadIdx.x == 0 && blockIdx.x == 0)
        printf("Hello from 2\n");
}

int main (int argc, char **argv) {
    hipStream_t streams[N];
    for(int i = 0; i < N; i++)
        hipStreamCreate(&streams[i]);
    printf("Start\n");
    Kernel1<<<128, 256, 0, streams[0]>>>();
    Kernel2<<<128, 256, 0, streams[1]>>>();
    printf("Finish\n");
	return 0;
}
